#include "hip/hip_runtime.h"
#include "HistogramGPU.cuh"
#include "Defines.h"
#include "Results.h"
#include <iostream>

#define MIN3(x,y,z)  ((y) <= (z) ? \
		((x) <= (y) ? (x) : (y)) \
		: \
		  ((x) <= (z) ? (x) : (z)))
#define MAX3(x,y,z)  ((y) >= (z) ? \
		((x) >= (y) ? (x) : (y)) \
		: \
		  ((x) >= (z) ? (x) : (z)))

inline void checkCuda(hipError_t e, string onde)
{
	if(e != hipSuccess)
		printf("Cuda error %d (%s): %s\n", e, onde.c_str(), hipGetErrorString(e));

}

__global__ void histograma_kernel_basic(unsigned char *img, int linhas, int colunas, int n, unsigned int *histo)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int zIndex = blockDim.z * blockIdx.z + threadIdx.z;

	int id = (yIndex + xIndex * colunas) + (linhas * colunas * zIndex) ;
	id = id*3;

	if(xIndex < linhas && yIndex < colunas && zIndex < n)
	{
		unsigned char r=img[id+2];
		atomicAdd(&(histo[r+(256*zIndex)]),1);
		//atomicAdd(&(histo[g+(256*3*zIndex)+256]),1);
		//atomicAdd(&(histo[b+(256*3*zIndex)+2*256]),1);
	}
}

__global__ void variance_kernel(unsigned int* histo, float* norm, int freq, int n)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

	int id = (xIndex + yIndex * 256) ;

	if(xIndex < 256 && yIndex < n)
	{
		float aux = __fdividef((float)histo[id],freq);
		norm[id] = __powf(aux-0.0625, 2);
	}
}

__global__ void hsv_kernel(unsigned char *img, int linhas, int colunas, int n)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int zIndex = blockDim.z * blockIdx.z + threadIdx.z;

	int id = (yIndex + xIndex * colunas) + (linhas * colunas * zIndex) ;
	id = id*3;

	if(xIndex < linhas && yIndex < colunas && zIndex < n)
	{
		unsigned char h,s,v;
		unsigned char r=img[id+2], g=img[id+1], b=img[id];
		unsigned char rgb_min=MIN3(r,g,b);
		unsigned char rgb_max=MAX3(r,g,b);

		unsigned char delMax = rgb_max - rgb_min;
		h = 0;
		s = 0;
		v = rgb_max;

		if(delMax == 0)
		{
			h = 0; s = 0;
		}
		else
		{
			s = delMax/255;
			if(rgb_max == r)
				h = ((g-b)/delMax)*60;
			else if(rgb_max == g)
				h = (2 + (b-r)/delMax)*60;
			else
				h = (4 + (r-g)/delMax)*60;
		}
		img[id] = h;
		img[id+1] = s;
		img[id+2] = v;
	}
}

__global__ void histograma_kernel_basic(unsigned char *img, int linhas, int colunas, unsigned int *histo)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

	int id = (yIndex + xIndex * colunas);

	id = id*3;

	if(xIndex < linhas && yIndex < colunas)
	{
		//printf("id:%d (%d-%d)\n", id, xIndex, yIndex);
		unsigned char r=img[id];//, g=img[id+1], b=img[id];
		atomicAdd(&(histo[r]),1);
		//atomicAdd(&(histo[g+256]),1);
		//atomicAdd(&(histo[b+2*256]),1);
	}
}

__global__ void continuous_kernel(cv::gpu::DevMem2D_<uchar4> mat, unsigned char *continuos) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	const uchar4& in = mat.ptr(y)[x];
	continuos[y*(mat.cols)+x                      ] = in.z;
	continuos[y*(mat.cols)+x+  (mat.cols*mat.rows)] = in.y;
	continuos[y*(mat.cols)+x+(2*mat.cols*mat.rows)] = in.x;
	//printf("[%d-%d] = (%d,%d,%d)\n", x, y, in.z, in.y, in.x);
}


__global__ void histo_kernel( unsigned char *buffer, long size, unsigned int *histo) {

	__shared__  unsigned int temp[256];
	temp[threadIdx.x+0] = 0;
	__syncthreads();

	int i = threadIdx.x + blockIdx.x * blockDim.x;

	//i = i*4;
	int stride = blockDim.x * gridDim.x;

	while (i < size) {
		atomicAdd( &temp[buffer[i]], 1 );
		i += stride;
	}

	__syncthreads();
	atomicAdd( &(histo[(threadIdx.x+0)]), temp[threadIdx.x+0] );

}


__global__ void variance_kernel(unsigned int* histo, float *norm, int freq){
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if(i < 256)
	{
		float aux = (float)histo[i]/freq;
		norm[i] = (aux-(1/256))*(aux-(1/256));
		//printf("%d = %f - %f * 2 = %f\n", i, aux, 1/256, (aux-(1/256))*(aux-(1/256)));
	}
}

__global__ void hsv_kernel(unsigned char *data, int linhas, int colunas)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

	int id = (yIndex + xIndex * colunas);

	if(xIndex < linhas && yIndex < colunas)
	{
		unsigned char h,s,v;
		unsigned char r=data[id], g=data[id+(linhas*colunas)], b=data[id+2*linhas*colunas];
		unsigned char rgb_min=MIN3(r,g,b);
		unsigned char rgb_max=MAX3(r,g,b);

		unsigned char delMax = rgb_max - rgb_min;
		h = 0;
		s = 0;
		v = rgb_max;

		if(delMax == 0)
		{
			h = 0; s = 0;
		}
		else
		{
			s = delMax/255;
			if(rgb_max == r)
				h = ((g-b)/delMax)*60;
			else if(rgb_max == g)
				h = (2 + (b-r)/delMax)*60;
			else
				h = (4 + (r-g)/delMax)*60;
		}
		data[id] = h;
		data[id+(linhas*colunas)] = s;
		data[id+2*(linhas*colunas)] = v;
	}
}

HistogramGPU::HistogramGPU()
{
	this->bins = 16;
	this->channels = 3;
}

HistogramGPU::HistogramGPU(vector<float> histogram, int bins, int idFrame, int chan, int freq)
: hist(histogram), idFrame(idFrame), bins(bins), channels(chan), freqTotal(freq)
{
}

vector<float> HistogramGPU::getHistogramNorm()
{
	vector<float> norm;
	for(int i=0; i<BINS; i++)
		norm.push_back(getHistPosNorm(i));
	return norm;
}

vector<HistogramGPU> FeaturesGPU::computeAllHist(unsigned char* images, int frameInicial, int qntFrames, int rows, int cols )
{
	cv::TickMeter timeLocal; double time = 0.0;

	timeLocal.reset(); timeLocal.start();

	vector<HistogramGPU> hists;
	int freqTotal = rows*cols;
	long size = freqTotal*3*qntFrames;
	int histsSize = 256*3*qntFrames;

	unsigned char* imagemGPU;
	unsigned int* histogramaGPU;
	unsigned int* histogramaGPUFinal;
	float* varianciasGPU;

	float* varianciasCPU = (float*)malloc(histsSize*sizeof(float));

	checkCuda(hipMalloc((void**)&imagemGPU, size*sizeof(unsigned char)),"hipMalloc imagemGPU failed");
	checkCuda(hipMalloc((void**)&histogramaGPU, histsSize*sizeof(unsigned int)),"hipMalloc histogramaGPU failed");
	checkCuda(hipMalloc((void**)&varianciasGPU, histsSize*sizeof(float)),"hipMalloc varianciasGPU failed");

	timeLocal.stop(); time += timeLocal.getTimeSec();

	checkCuda(hipMemcpy(imagemGPU, images, size*sizeof(unsigned char), hipMemcpyHostToDevice),"hipMemcpy imagemGPU failed");
	checkCuda(hipMemset(histogramaGPU, 0, histsSize*sizeof(unsigned int)),"hipMemset histogramaGPU failed");
	checkCuda(hipMemset(varianciasGPU, 0, histsSize*sizeof(float)),"hipMemset varianciasGPU failed");

	timeLocal.reset(); timeLocal.start();

	dim3 dimBlock(16,16,4);
	int blocksInX = (rows+16-1)/16;
	int blocksInY = (cols+16-1)/16;
	int blocksInZ = (qntFrames+4-1)/4;
	dim3 dimGrid(blocksInX, blocksInY, blocksInZ);

	histograma_kernel_basic<<<dimGrid, dimBlock>>>(imagemGPU, rows, cols, qntFrames, histogramaGPU);
	checkCuda(hipGetLastError(), "last erro");
	checkCuda(hipDeviceSynchronize(), "cudaSynchronize failed");

	variance_kernel<<<dimGrid, dimBlock>>>(histogramaGPU, varianciasGPU, freqTotal, qntFrames);
	checkCuda(hipGetLastError(), "last erro");
	checkCuda(hipDeviceSynchronize(), "cudaSynchronize failed");

	timeLocal.stop(); time += timeLocal.getTimeSec();

	checkCuda(hipMemcpy(varianciasCPU, varianciasGPU, histsSize * sizeof(float), hipMemcpyDeviceToHost),"hipMemcpy varianciasGPU failed");

	hipFree(histogramaGPU);
	hipFree(varianciasGPU);

	timeLocal.reset(); timeLocal.start();

	hsv_kernel<<<dimGrid, dimBlock>>>(imagemGPU, rows, cols, qntFrames);
	checkCuda(hipGetLastError(), "last erro");
	checkCuda(hipDeviceSynchronize(), "cudaSynchronize failed");

	timeLocal.stop(); time += timeLocal.getTimeSec();

	checkCuda(hipMalloc((void**)&histogramaGPUFinal, histsSize*sizeof(unsigned int)),"hipMalloc histogramaGPUFinal failed");
	checkCuda(hipMemset(histogramaGPUFinal, 0, histsSize*sizeof(unsigned int)),"hipMemset histogramaGPUFinal failed");

	timeLocal.reset(); timeLocal.start();

	histograma_kernel_basic<<<dimGrid, dimBlock>>>(imagemGPU, rows, cols, qntFrames, histogramaGPUFinal);
	checkCuda(hipGetLastError(), "last erro");
	checkCuda(hipDeviceSynchronize(), "cudaSynchronize failed");

	timeLocal.stop(); time += timeLocal.getTimeSec();

	unsigned int* histogramaCPUFinal = (unsigned int*)malloc(histsSize*sizeof(unsigned int));
	checkCuda(hipMemcpy(histogramaCPUFinal, histogramaGPUFinal, histsSize * sizeof(float), hipMemcpyDeviceToHost),"hipMemcpy histogramaGPUFinal failed");

	timeLocal.reset(); timeLocal.start();

	int idInicial = frameInicial;
	for(int i=0; i<qntFrames; i++)
	{
		idInicial++;

		float desvios  = 0.0;
		for(int j=0; j<256; j++)
			desvios += varianciasCPU[(i*256)+j];
		float d = sqrt(desvios/16.0);
		if(d > 0.23)
			continue;

		vector<float> hist(BINS,0.0);
		for(int j=0; j<256; j++)
			hist[j/BINS] += (histogramaCPUFinal[(i*256)+j]);

		hists.push_back(HistogramGPU(hist, BINS, idInicial, 3, freqTotal));
	}

	timeLocal.stop(); time += timeLocal.getTimeSec();

	Results *result;
	result = Results::getInstance();
	result->setFeatExtractionGpuCopy(time);

	hipFree(imagemGPU);
	hipFree(histogramaGPUFinal);

	free(varianciasCPU);
	free(histogramaCPUFinal);

	return hists;
}




HistogramGPU FeaturesGPU::computeOneHist(cv::gpu::GpuMat img, int idFrame, int rows, int cols)
{
	//cout << "compute one hist" << endl;
	cv::TickMeter timeLocal; double time = 0.0;


	timeLocal.reset(); timeLocal.start();
	HistogramGPU hist;

	//cuda computations
	hipError_t cudaStatus;

	int freqTotal = rows*cols;
	long size = freqTotal*3;
	int histsSize = 256;

	//aloca memoria para imagens e copia para device
	unsigned char* imagemGPU;
	cudaStatus = hipMalloc((void**)&imagemGPU, size*sizeof(unsigned char));
	if (cudaStatus != hipSuccess) cerr << "hipMalloc imagemGPU failed " <<  hipGetErrorString(cudaStatus) << endl;
	//cudaStatus = hipMemcpy(imagemGPU, image, size*sizeof(unsigned char), hipMemcpyHostToDevice);
	//if (cudaStatus != hipSuccess) cerr << "hipMemcpy imagemGPU failed " <<  hipGetErrorString(cudaStatus) << endl;

	//aloca memoria histograma
	unsigned int* histogramaGPU;
	cudaStatus = hipMalloc((void**)&histogramaGPU, histsSize*sizeof(unsigned int));
	if (cudaStatus != hipSuccess) cerr << "hipMalloc histogramaGPU failed " <<  hipGetErrorString(cudaStatus) << endl;


	timeLocal.stop(); time += timeLocal.getTimeSec();

	cudaStatus = hipMemset(histogramaGPU, 0, histsSize*sizeof(unsigned int));
	if (cudaStatus != hipSuccess) cerr << "hipMemset histogramaGPU failed " <<  hipGetErrorString(cudaStatus) << endl;

	timeLocal.reset(); timeLocal.start();

	//escolhe tamanhos blocos e grid
	dim3 dimBlock(16,16);
	int blocksInX = (rows+16-1)/16;
	int blocksInY = (cols+16-1)/16;
	dim3 dimGrid(blocksInY, blocksInX);

	continuous_kernel<<<dimGrid,dimBlock>>>(img,imagemGPU);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) cerr << "failed:" << hipGetErrorString(cudaStatus) << endl;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) cerr << "cudaSynchronize failed" << endl;

	img.release();

	//chama kernel histograma
	histo_kernel<<<4, 256>>>(imagemGPU, rows*cols, histogramaGPU);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) cerr << "failed:" << hipGetErrorString(cudaStatus) << endl;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) cerr << "cudaSynchronize failed" << endl;

	timeLocal.stop(); time += timeLocal.getTimeSec();

	unsigned int* histTeste = (unsigned int*)malloc(histsSize*sizeof(unsigned int));
	cudaStatus = hipMemcpy(histTeste, histogramaGPU, histsSize * sizeof(float), hipMemcpyDeviceToHost);

	//cout << "frame: " << idFrame <<  endl;
	//for(int i=0; i<histsSize; i++)
	//	cout << (int)histTeste[i] << " ";
	//cout << endl;


	//exit(1);
	timeLocal.reset(); timeLocal.start();

	//aloca memoria para calcular variancias
	float* varianciasGPU;
	cudaStatus = hipMalloc((void**)&varianciasGPU, 256*sizeof(float));
	if (cudaStatus != hipSuccess) cerr << "hipMalloc varianciasGPU failed " <<  hipGetErrorString(cudaStatus) << endl;
	//cudaStatus = hipMemset(varianciasGPU, 0, 256*sizeof(float));
	//if (cudaStatus != hipSuccess) cerr << "hipMemset varianciasGPU failed " <<  hipGetErrorString(cudaStatus) << endl;


	//chame kernel variancias
	variance_kernel<<<4, 256>>>(histogramaGPU, varianciasGPU, freqTotal);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) cerr << "failed:" << hipGetErrorString(cudaStatus) << endl;
	hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) cerr << "cudaSynchronize failed" << endl;

	timeLocal.stop(); time += timeLocal.getTimeSec();

	//copia variancias para host
	float* varianciasCPU = (float*)malloc(256*sizeof(float));
	cudaStatus = hipMemcpy(varianciasCPU, varianciasGPU, 256 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) cerr << "hipMemcpy varianciasGPU failed " <<  hipGetErrorString(cudaStatus) << endl;

	timeLocal.reset(); timeLocal.start();

	hipFree(histogramaGPU);
	hipFree(varianciasGPU);

	//elimina frames inexpressivos
	float desvio = 0.0;

	for(int j=0; j<256; j++)
		desvio += varianciasCPU[j];

	float d = sqrt(desvio);
	if(d > 0.5)
	{
		return hist;
	}


	//kernel RGB to HSV
	hsv_kernel<<<dimGrid, dimBlock>>>(imagemGPU, rows, cols);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) cerr << "failed:" << hipGetErrorString(cudaStatus) << endl;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) cerr << "cudaSynchronize failed" << endl;


	//aloca memoria histograma HSV
	unsigned int* histogramaGPUFinal;
	cudaStatus = hipMalloc((void**)&histogramaGPUFinal, histsSize*sizeof(unsigned int));
	if (cudaStatus != hipSuccess) cerr << "hipMalloc histogramaGPUFinal failed " <<  hipGetErrorString(cudaStatus) << endl;

	timeLocal.stop(); time += timeLocal.getTimeSec();

	cudaStatus = hipMemset(histogramaGPUFinal, 0, histsSize*sizeof(unsigned int));
	if (cudaStatus != hipSuccess) cerr << "hipMemset histogramaGPUFinal failed " <<  hipGetErrorString(cudaStatus) << endl;;

	timeLocal.reset(); timeLocal.start();


	//kernel histograma
	histo_kernel<<<4, 256>>>(imagemGPU, rows*cols, histogramaGPUFinal);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) cerr << "failed:" << hipGetErrorString(cudaStatus) << endl;
	hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) cerr << "cudaSynchronize failed" << endl;

	timeLocal.stop(); time += timeLocal.getTimeSec();

	//copia historgama final para host
	unsigned int* histogramaCPUFinal = (unsigned int*)malloc(histsSize*sizeof(unsigned int));
	cudaStatus = hipMemcpy(histogramaCPUFinal, histogramaGPUFinal, histsSize * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) cerr << "hipMemcpy histogramaGPUFinal failed " <<  hipGetErrorString(cudaStatus) << endl;

	timeLocal.reset(); timeLocal.start();

	vector<float> histF(BINS,0.0);
	for(int j=0; j<256; j++)
		histF[j/BINS] += (histogramaCPUFinal[j]);

	//for(int i=0; i<histF.size(); i++)
	//	cout << histF[i] << " ";
	//cout << endl;

	hist.setHistogram(histF);
	hist.setBins(BINS);
	hist.setIdFrame(idFrame);
	hist.setFreqTotal(freqTotal);
	hist.setChannels(3);

	//cout << "size: " << hist.getHistogram().size() << endl;
	hipFree(imagemGPU);
	hipFree(histogramaGPUFinal);

	free(varianciasCPU);
	free(histogramaCPUFinal);

	timeLocal.stop(); time += timeLocal.getTimeSec();


	Results *result;
	result = Results::getInstance();
	result->setFeatExtractionGpuCopy(time);

	if (cudaStatus != hipSuccess)
		exit(1);

	return hist;
}

